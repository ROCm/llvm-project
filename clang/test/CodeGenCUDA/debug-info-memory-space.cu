
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple nvptx-unknown-unknown -emit-llvm -fcuda-is-device -debug-info-kind=limited -gheterogeneous-dwarf=diexpr -o - %s | FileCheck %s
// CHECK-DAG: !DIGlobalVariable(name: "GlobalShared", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true, memorySpace: DW_MSPACE_LLVM_group)
// CHECK-DAG: !DIGlobalVariable(name: "GlobalDevice", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true, memorySpace: DW_MSPACE_LLVM_global)
// CHECK-DAG: !DIGlobalVariable(name: "GlobalConstant", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true, memorySpace: DW_MSPACE_LLVM_constant)
// CHECK-DAG: !DIGlobalVariable(name: "FuncVarShared", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: true, isDefinition: true, memorySpace: DW_MSPACE_LLVM_group)
// CHECK-DAG: !DILocalVariable(name: "FuncVar", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}})

// CHECK-DAG: !DILocalVariable(name: "FuncVarSharedPointer", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: ![[DEVICE_PTR:[0-9]+]])
// CHECK-DAG: !DILocalVariable(name: "FuncVarPointer", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: ![[DEVICE_PTR:[0-9]+]])
// CHECK-DAG: ![[DEVICE_PTR]] = !DIDerivedType(tag: DW_TAG_pointer_type, baseType: !{{[0-9]+}}, size: {{[0-9]+}})

#define __device__ __attribute__((device))
#define __shared__ __attribute__((shared))
#define __constant__ __attribute__((constant))

__shared__ int GlobalShared;
__device__ int GlobalDevice;
__constant__ int GlobalConstant;

__device__ void kernel1(int Arg) {
  __shared__ int FuncVarShared;
  int FuncVar;

  auto *FuncVarSharedPointer = &FuncVarShared;
  auto *FuncVarPointer = &FuncVar;
}
